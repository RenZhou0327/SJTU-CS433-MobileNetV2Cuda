#include "hip/hip_runtime.h"
#include "layers.cuh"

__global__ void add_bias_relu6(float* WX, float *B, int out_c, int out_shape) 
{
    // thread_j: [0, 122 * 32)
    int thread_j = blockIdx.x * blockDim.x + threadIdx.x;
    // thread_i: [0, 122)
    int thread_i = blockIdx.y;
    int num_id = thread_i * out_c * out_shape + thread_j;
    int b_id = num_id / (out_shape * out_shape);
    WX[num_id] += B[b_id];
    // RELU6
    WX[num_id] = max(0.0, WX[num_id]);
    WX[num_id] = min(6.0, WX[num_id]);
    // printf("%d %d\n", thread_i, thread_j);
}


__global__ void img2col(float *imgs, float *cols, int in_shape, int out_shape, int k_shape, int in_c, int s, int p) {
    int thread_j = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_i = blockIdx.y * blockDim.y + threadIdx.y;
    int cols_id = thread_i * (out_shape * out_shape) + thread_j;
    
    int row_idx = cols_id / (out_shape * out_shape);
    int col_idx = cols_id % (out_shape * out_shape);
    
    int c_idx = row_idx / (k_shape * k_shape);
    int i_idx = (row_idx / k_shape) % k_shape + (col_idx / out_shape) * s - p;
    int j_idx = row_idx % in_c + (col_idx % out_shape) * s - p;
    // if (d_idx == 2) {
    //     printf("%d %d %d %d\n", cols_id, d_idx, i_idx, j_idx);
    // }
    if (i_idx >= 0 && j_idx >= 0) {
        int img_idx = c_idx * (in_shape * in_shape) + i_idx * in_shape + j_idx;
        cols[cols_id] = imgs[img_idx];
    }
}


// __global__ void add_bias_kernel(float* WX, float *B, int out_c, int out_shape) {

    
// }


void mat_multiple(float *A, float *B, float* C, int m, int k, int n) {
	hipblasStatus_t status;
	hipblasHandle_t handle;

    const float al = 1.0f, bt = 0.0f;
    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &al, B, n, A, k, &bt, C, n);
}





void conv2d(float* in_tensor, float** out_tensor_p, float* w, float* b, int in_shape, int in_c, int k_shape, int out_c, int stride, int pad)
{
    int out_shape = int((in_shape + 2 * pad - k_shape) / stride) + 1;
    printf("out shape: %d\n", out_shape);

    float *in_cols = NULL;
    int threadNum = k_shape * k_shape * in_c * out_shape * out_shape;
    hipError_t err = hipSuccess;
    err = hipMalloc((void**)&in_cols, threadNum * sizeof(float));
    assert(err == hipSuccess);
    
    int bIndx = ceil(out_shape * out_shape / 32.0), bIndy = in_c;
    int tIndx = 32, tIndy = k_shape * k_shape;
    // printf("%d %d %d %d %d\n", threadNum, bIndx, bIndy, tIndx, tIndy);
    // exit(0);
    dim3 gDim1(bIndx, bIndy);
    // !!! 特别注意, tIndx * tIndy得小于1024, 否则出错无结果!!!
    dim3 bDim1(tIndx, tIndy);
    img2col<<<gDim1, bDim1>>>(in_tensor, in_cols, in_shape, out_shape, k_shape, in_c, stride, pad);
    hipFree(in_tensor);

    // // Just for Test:
    // float *temp = (float*) malloc(threadNum * sizeof(float));
    // err = hipMemcpy(temp, in_cols, threadNum * sizeof(float), hipMemcpyDeviceToHost);
    // assert(err == hipSuccess);
    // printf("%f\n", temp[400868]);
    // for (int i = 0; i < 10; ++i) {
    //     printf("%f ", temp[10388]);
    // }
    // printf("\n");
    // exit(0);

    float *out_tensor = NULL;
    int out_lens = out_c * out_shape * out_shape;
    int mat_m = out_c, mat_k = in_c * k_shape * k_shape, mat_n = out_shape * out_shape;
    err = hipMalloc((void**)&out_tensor, out_lens * sizeof(float));
    assert(err == hipSuccess);
    mat_multiple(w, in_cols, out_tensor, mat_m, mat_k, mat_n);
    err = hipFree(w);
    assert(err == hipSuccess);
    err = hipFree(in_cols);
    assert(err == hipSuccess);

    printf("here\n");
    printf("%d %d\n", out_c, out_shape);
    dim3 gDim2(out_shape, out_shape);
    dim3 bDim2(out_c, 1);
    add_bias_relu6<<<gDim2, bDim2>>>(out_tensor, b, out_c, out_shape);
    hipFree(b);

    // Just for Test:
    float *temp = (float*) malloc(out_lens * sizeof(float));
    err = hipMemcpy(temp, out_tensor, out_lens * sizeof(float), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    // printf("%f\n", temp[416288]);
    FILE *test_file = fopen("tmpfiles/317_relu.txt", "w");
    for (int i = 0; i < out_lens; ++i) {
        fprintf(test_file, "%f ", temp[i]);
    }
    fprintf(test_file, "\n");
    fclose(test_file);
    exit(0);
};

void relu6() {};
void depth_wise_conv() {};
void point_wise_conv() {};
void add_layer() {};
void avg_pool() {};
void linear_layer() {};
